#include "hip/hip_runtime.h"
/*
 * Copyright 2021 PlenOctree Authors
 * Copyright 2022 Artemis Authors
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdint>
#include <vector>
#include "common.cuh"
#include "data_spec_packed.cuh"

namespace {

// Automatically choose number of CUDA threads based on HW CUDA kernel count
int cuda_n_threads = -1;
__host__ void auto_cuda_threads() {
    if (~cuda_n_threads) return;
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);
    const int n_cores = get_sp_cores(dev_prop);
    // Optimize number of CUDA threads per block
    if (n_cores < 2048) {
        cuda_n_threads = 256;
    } if (n_cores < 8192) {
        cuda_n_threads = 512;
    } else {
        cuda_n_threads = 512;
    }
}

namespace device {
// SH Coefficients from https://github.com/google/spherical-harmonics
__device__ __constant__ const float C0 = 0.28209479177387814;
__device__ __constant__ const float C1 = 0.4886025119029199;
__device__ __constant__ const float C2[] = {
    1.0925484305920792,
    -1.0925484305920792,
    0.31539156525252005,
    -1.0925484305920792,
    0.5462742152960396
};

__device__ __constant__ const float C3[] = {
    -0.5900435899266435,
    2.890611442640554,
    -0.4570457994644658,
    0.3731763325901154,
    -0.4570457994644658,
    1.445305721320277,
    -0.5900435899266435
};

__device__ __constant__ const float C4[] = {
    2.5033429417967046,
    -1.7701307697799304,
    0.9461746957575601,
    -0.6690465435572892,
    0.10578554691520431,
    -0.6690465435572892,
    0.47308734787878004,
    -1.7701307697799304,
    0.6258357354491761,
};


template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _norm(
                scalar_t* dir) {
    return sqrtf(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
}

template<typename scalar_t>
__host__ __device__ __inline__ static void _normalize(
                scalar_t* dir) {
    scalar_t norm = _norm(dir);
    dir[0] /= norm; dir[1] /= norm; dir[2] /= norm;
}

template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _dot3(
        const scalar_t* __restrict__ u,
        const scalar_t* __restrict__ v) {
    return u[0] * v[0] + u[1] * v[1] + u[2] * v[2];
}


// Calculate basis functions depending on format, for given view directions
template <typename scalar_t>
__device__ __inline__ void maybe_precalc_basis(
    const int format,
    const int basis_dim,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        extra,
    const scalar_t* __restrict__ dir,
    scalar_t* __restrict__ out) {
    switch(format) {
        case FORMAT_ASG:
            {
                // UNTESTED ASG
                for (int i = 0; i < basis_dim; ++i) {
                    const auto& ptr = extra[i];
                    scalar_t S = _dot3(dir, &ptr[8]);
                    scalar_t dot_x = _dot3(dir, &ptr[2]);
                    scalar_t dot_y = _dot3(dir, &ptr[5]);
                    out[i] = S * expf(-ptr[0] * dot_x * dot_x
                                      -ptr[1] * dot_y * dot_y) / basis_dim;
                }
            }  // ASG
            break;
        case FORMAT_SG:
            {
                for (int i = 0; i < basis_dim; ++i) {
                    const auto& ptr = extra[i];
                    out[i] = expf(ptr[0] * (_dot3(dir, &ptr[1]) - 1.f)) / basis_dim;
                }
            }  // SG
            break;
        case FORMAT_SH:
            {
                out[0] = C0;
                const scalar_t x = dir[0], y = dir[1], z = dir[2];
                const scalar_t xx = x * x, yy = y * y, zz = z * z;
                const scalar_t xy = x * y, yz = y * z, xz = x * z;
                switch (basis_dim) {
                    case 25:
                        out[16] = C4[0] * xy * (xx - yy);
                        out[17] = C4[1] * yz * (3 * xx - yy);
                        out[18] = C4[2] * xy * (7 * zz - 1.f);
                        out[19] = C4[3] * yz * (7 * zz - 3.f);
                        out[20] = C4[4] * (zz * (35 * zz - 30) + 3);
                        out[21] = C4[5] * xz * (7 * zz - 3);
                        out[22] = C4[6] * (xx - yy) * (7 * zz - 1.f);
                        out[23] = C4[7] * xz * (xx - 3 * yy);
                        out[24] = C4[8] * (xx * (xx - 3 * yy) - yy * (3 * xx - yy));
                        [[fallthrough]];
                    case 16:
                        out[9] = C3[0] * y * (3 * xx - yy);
                        out[10] = C3[1] * xy * z;
                        out[11] = C3[2] * y * (4 * zz - xx - yy);
                        out[12] = C3[3] * z * (2 * zz - 3 * xx - 3 * yy);
                        out[13] = C3[4] * x * (4 * zz - xx - yy);
                        out[14] = C3[5] * z * (xx - yy);
                        out[15] = C3[6] * x * (xx - 3 * yy);
                        [[fallthrough]];
                    case 9:
                        out[4] = C2[0] * xy;
                        out[5] = C2[1] * yz;
                        out[6] = C2[2] * (2.0 * zz - xx - yy);
                        out[7] = C2[3] * xz;
                        out[8] = C2[4] * (xx - yy);
                        [[fallthrough]];
                    case 4:
                        out[1] = -C1 * y;
                        out[2] = C1 * z;
                        out[3] = -C1 * x;
                }
            }  // SH
            break;

        default:
            // Do nothing
            break;
    }  // switch
}

template <typename scalar_t>
__device__ __inline__ scalar_t _get_delta_scale(
    const scalar_t* __restrict__ scaling,
    scalar_t* __restrict__ dir) {
    dir[0] *= scaling[0];
    dir[1] *= scaling[1];
    dir[2] *= scaling[2];
    scalar_t delta_scale = 1.f / _norm(dir);
    dir[0] *= delta_scale;
    dir[1] *= delta_scale;
    dir[2] *= delta_scale;
    return delta_scale;
}

template <typename scalar_t>
__device__ __inline__ void _dda_unit(
        const scalar_t* __restrict__ cen,
        const scalar_t* __restrict__ invdir,
        scalar_t* __restrict__ tmin,
        scalar_t* __restrict__ tmax) {
    // Perform DDA for 1 iteration on a unit cube
    scalar_t t1, t2;
    *tmin = 0.0f;
    *tmax = 1e9f;
#pragma unroll
    for (int i = 0; i < 3; ++i) {
        t1 = - cen[i] * invdir[i];
        t2 = t1 +  invdir[i];
        *tmin = max(*tmin, min(t1, t2));
        *tmax = min(*tmax, max(t1, t2));
    }
}


template <typename scalar_t>
__device__ __inline__ void trace_ray(
        PackedTreeSpec<scalar_t>& __restrict__ tree,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.features.size(1);
    const int out_data_dim = out.size(0) - 1;

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = opt.background_brightness;
        }
        out[out_data_dim] = 0;
        return;
    } else {
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        scalar_t pos[3];
        scalar_t basis_fn[25];
        maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim,
                tree.extra_data, ray.vdir, basis_fn);

        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        scalar_t cube_sz;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j]; 
            }

            int64_t node_id;
            int32_t* data_idx_ptr = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz, tree.weight_accum != nullptr ? &node_id : nullptr);
            
            scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);
            
            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + opt.step_size;
            const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;
            // printf("data_idx: %d, cub_size: %f, feature_size: %d, sigma: %f, delta: %f, data_dim: %d\n", *data_idx_ptr, cube_sz, tree.features.size(0), sigma, delta_t, data_dim);
            if (sigma > opt.sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                const scalar_t weight = light_intensity * (1.f - att);
                
                if (tree.transformation_matrices.size(0) != 0) {
                    scalar_t ray_dir[3] = {ray.vdir[0], ray.vdir[1], ray.vdir[2]};
                    ray_dir[0] = tree.transformation_matrices[*data_idx_ptr][0][0] * ray.vdir[0] + tree.transformation_matrices[*data_idx_ptr][0][1] * ray.vdir[1] + tree.transformation_matrices[*data_idx_ptr][0][2] * ray.vdir[2];
                    ray_dir[1] = tree.transformation_matrices[*data_idx_ptr][1][0] * ray.vdir[0] + tree.transformation_matrices[*data_idx_ptr][1][1] * ray.vdir[1] + tree.transformation_matrices[*data_idx_ptr][1][2] * ray.vdir[2];
                    ray_dir[2] = tree.transformation_matrices[*data_idx_ptr][2][0] * ray.vdir[0] + tree.transformation_matrices[*data_idx_ptr][2][1] * ray.vdir[1] + tree.transformation_matrices[*data_idx_ptr][2][2] * ray.vdir[2];
                    
                    maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim,
                        tree.extra_data, ray_dir, basis_fn);
                }

                if (opt.format != FORMAT_RGBA) {
                    for (int t = 0; t < out_data_dim; ++ t) {
                        int off = t * opt.basis_dim;
                        scalar_t tmp = 0.0;
                        for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                            tmp += basis_fn[i] * tree_val[off + i];
                        }
                        out[t] += weight / (1.0 + expf(-tmp));
                    }
                } else {
                    for (int j = 0; j < out_data_dim; ++j) {
                        out[j] += weight / (1.0 + expf(-tree_val[j]));
                    }
                }
                light_intensity *= att;

                if (tree.weight_accum != nullptr) {
                    tree.weight_accum[node_id] += weight;
                }

                if (light_intensity <= opt.stop_thresh) {
                    // Full opacity, stop
                    scalar_t scale = 1.0 / (1.0 - light_intensity);
                    for (int j = 0; j != out_data_dim; ++j) out[j] *= scale;
                    out[out_data_dim] = 1 - light_intensity;
                    return;
                }
            }
            t += delta_t;
        }
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] += light_intensity * opt.background_brightness;
        }
        out[out_data_dim] = 1 - light_intensity;
    }
}

template <typename scalar_t>
__device__ __inline__ void trace_ray_backward(
    PackedTreeSpec<scalar_t>& __restrict__ tree,
    const torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t>
        grad_output,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        grad_data_out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.features.size(1);
    const int out_data_dim = grad_output.size(0) - 1;

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        scalar_t pos[3];
        scalar_t basis_fn[25];
        maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim, tree.extra_data,
                ray.vdir, basis_fn);

        scalar_t accum = 0.0;
        scalar_t light_intensity_ray = 0.0;
        // PASS 1
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];

                int32_t* data_idx_ptr = query_single_from_root<scalar_t>(
                        tree.data, tree.child, pos, &cube_sz);
                scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;

                if (sigma > 0.0) {
                    // Reuse offset on gradient
                    const int64_t curr_leaf_offset = tree_val - tree.features.data();
                    scalar_t* grad_tree_val = grad_data_out.data() + curr_leaf_offset;

                    if (tree.transformation_matrices.size(0) != 0) {
                        scalar_t ray_dir[3] = {ray.vdir[0], ray.vdir[1], ray.vdir[2]};
                        ray_dir[0] = tree.transformation_matrices[*data_idx_ptr][0][0] * ray.vdir[0] + tree.transformation_matrices[*data_idx_ptr][0][1] * ray.vdir[1] + tree.transformation_matrices[*data_idx_ptr][0][2] * ray.vdir[2];
                        ray_dir[1] = tree.transformation_matrices[*data_idx_ptr][1][0] * ray.vdir[0] + tree.transformation_matrices[*data_idx_ptr][1][1] * ray.vdir[1] + tree.transformation_matrices[*data_idx_ptr][1][2] * ray.vdir[2];
                        ray_dir[2] = tree.transformation_matrices[*data_idx_ptr][2][0] * ray.vdir[0] + tree.transformation_matrices[*data_idx_ptr][2][1] * ray.vdir[1] + tree.transformation_matrices[*data_idx_ptr][2][2] * ray.vdir[2];
                        
                        maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim,
                            tree.extra_data, ray_dir, basis_fn);
                    }

                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    scalar_t total_color = 0.f;
                    if (opt.format != FORMAT_RGBA) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * tree_val[off + i];
                            }
                            const scalar_t sigmoid = 1.0 / (1.0 + expf(-tmp));
                            const scalar_t grad_sigmoid = sigmoid * (1.0 - sigmoid);
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                const scalar_t toadd = weight * basis_fn[i] *
                                    grad_sigmoid * grad_output[t];
                                atomicAdd(&grad_tree_val[off + i],
                                        toadd);
                            }
                            total_color += sigmoid * grad_output[t];
                        }
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            const scalar_t sigmoid = 1.0 / (1.0 + expf(-tree_val[j]));
                            const scalar_t toadd = weight * sigmoid * (
                                    1.f - sigmoid) * grad_output[j];
                            atomicAdd(&grad_tree_val[j], toadd);
                            total_color += sigmoid * grad_output[j];
                        }
                    }
                    light_intensity *= att;
                    accum += weight * total_color;
                }
                t += delta_t;
            }
            scalar_t total_grad = 0.f;
            for (int j = 0; j < out_data_dim; ++j)
                total_grad += grad_output[j];
            accum += light_intensity * opt.background_brightness * total_grad;
            light_intensity_ray = light_intensity;
        }
        // PASS 2
        {
            // scalar_t accum_lo = 0.0;
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];
                int32_t* data_idx_ptr = query_single_from_root<scalar_t>(
                        tree.data, tree.child, pos, &cube_sz);
                scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];
                
                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;

                if (sigma > 0.0) {
                    // Reuse offset on gradient
                    const int64_t curr_leaf_offset = tree_val - tree.features.data();
                    scalar_t* grad_tree_val = grad_data_out.data() + curr_leaf_offset;

                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    scalar_t total_color = 0.f;
                    if (opt.format != FORMAT_RGBA) {
                        for (int t = 0; t < out_data_dim; ++ t) {
                            int off = t * opt.basis_dim;
                            scalar_t tmp = 0.0;
                            for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                                tmp += basis_fn[i] * tree_val[off + i];
                            }
                            total_color += 1.0 / (1.0 + expf(-tmp)) * grad_output[t];
                        }
                    } else {
                        for (int j = 0; j < out_data_dim; ++j) {
                            total_color += 1.0 / (1.0 + expf(-tree_val[j])) * grad_output[j];
                        }
                    }
                    light_intensity *= att;
                    accum -= weight * total_color;
                    // atomicAdd(
                    //        &grad_tree_val[out_data_dim],
                    //        delta_t * delta_scale * (
                    //            total_color * light_intensity - accum)
                    //        );
                    atomicAdd(
                            &grad_tree_val[data_dim - 1],
                            delta_t * delta_scale * (
                                total_color * light_intensity - accum) + delta_t * delta_scale * grad_output[out_data_dim] * light_intensity_ray
                            );
                }
                t += delta_t;
            }
        }
    }
}


template <typename scalar_t>
__device__ __inline__ void opacity_trace_ray(
        PackedTreeSpec<scalar_t>& __restrict__ tree,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.features.size(1);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        out[0] = 0.f;
        return;
    } else {
        out[0] = 0.f;
        scalar_t pos[3];
        
        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        scalar_t cube_sz;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
            }

            int64_t node_id;
            int32_t* data_idx_ptr = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz, tree.weight_accum != nullptr ? &node_id : nullptr);
            
            scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);
            
            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + opt.step_size;
            const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;
            if (sigma > opt.sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                light_intensity *= att;
                if (light_intensity <= opt.stop_thresh) {
                    // Full opacity, stop
                    out[0] = 1 - light_intensity;
                    return;
                }
            }
            t += delta_t;
        }
        out[0] = 1 - light_intensity;
    }
}

template <typename scalar_t>
__device__ __inline__ void opacity_trace_ray_backward(
    PackedTreeSpec<scalar_t>& __restrict__ tree,
    const torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t>
        grad_output,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        grad_data_out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.features.size(1);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        scalar_t pos[3];

        scalar_t accum = 0.0;
        scalar_t light_intensity_ray = 0.0;
        // PASS 1
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];

                int32_t* data_idx_ptr = query_single_from_root<scalar_t>(
                        tree.data, tree.child, pos, &cube_sz);
                scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;

                if (sigma > 0.0) {
                    // Reuse offset on gradient
                    att = expf(-delta_t * sigma * delta_scale);
                    light_intensity *= att;
                }
                t += delta_t;
            }
            light_intensity_ray = light_intensity;
        }
        // PASS 2
        {
            // scalar_t accum_lo = 0.0;
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];
                int32_t* data_idx_ptr = query_single_from_root<scalar_t>(
                        tree.data, tree.child, pos, &cube_sz);
                scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];
                
                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;

                if (sigma > 0.0) {
                    // Reuse offset on gradient
                    const int64_t curr_leaf_offset = tree_val - tree.features.data();
                    scalar_t* grad_tree_val = grad_data_out.data() + curr_leaf_offset;
                    att = expf(-delta_t * sigma * delta_scale);
                    atomicAdd(
                            &grad_tree_val[data_dim - 1],
                            delta_t * delta_scale * grad_output[0] * light_intensity_ray
                            );
                }
                t += delta_t;
            }
        }
    }
}


template <typename scalar_t>
__global__ void render_ray_kernel(
        PackedTreeSpec<scalar_t> tree,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        out) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    // printf("matrices: %d\n", tree.transformation_matrices.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        out[tid]);
}


template <typename scalar_t>
__global__ void render_ray_backward_kernel(
    PackedTreeSpec<scalar_t> tree,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        grad_output,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        grad_data_out
        ) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    trace_ray_backward<scalar_t>(
        tree,
        grad_output[tid],
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        grad_data_out);
}


// motion feature ray tracing 
template <typename scalar_t>
__device__ __inline__ void motion_trace_ray(
        PackedTreeSpec<scalar_t>& __restrict__ tree,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out, 
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> depth_out,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> hit_point_out,
        torch::TensorAccessor<int64_t, 1, torch::RestrictPtrTraits, int32_t> data_idx_out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int data_dim = tree.features.size(1);
    const int out_data_dim = out.size(0);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        depth_out[0] = 0.f;
        hit_point_out[0] = 0.f;
        hit_point_out[1] = 0.f;
        hit_point_out[2] = 0.f;
        return;
    } else {
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        scalar_t pos[3];

        scalar_t t = tmin;
        scalar_t cube_sz;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
            }

            int64_t node_id;
            int32_t* data_idx_ptr = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz, tree.weight_accum != nullptr ? &node_id : nullptr);
            
            scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];

            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);
            
            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + opt.step_size;
            const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;
            // printf("data_idx: %d, cub_size: %f, feature_size: %d, sigma: %f, delta: %f, data_dim: %d\n", *data_idx_ptr, cube_sz, tree.features.size(0), sigma, delta_t, data_dim);
            if (sigma > opt.sigma_thresh) {
                transform_coord_world<scalar_t>(pos, tree.offset, tree.scaling);

                hit_point_out[0] = pos[0];
                hit_point_out[1] = pos[1];
                hit_point_out[2] = pos[2];

                depth_out[0] = t * delta_scale;
                scalar_t dis[3];
                for (int i = 0 ; i != out_data_dim ; ++i) {
                    dis[0] = pos[0] - tree.extra_data[i][0];
                    dis[1] = pos[1] - tree.extra_data[i][1];
                    dis[2] = pos[2] - tree.extra_data[i][2];
                    out[i] = _norm(dis);
                }
                data_idx_out[0] = *data_idx_ptr;

                return;
            }
            t += delta_t;
        }
    }
}

// depth ray tracing 
template <typename scalar_t>
__device__ __inline__ void depth_trace_ray(
        PackedTreeSpec<scalar_t>& __restrict__ tree,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> depth_out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int data_dim = tree.features.size(1);

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        depth_out[0] = 0.f; 
        return;
    } else {
        depth_out[0] = 0.f; 
        scalar_t pos[3];

        scalar_t t = tmin;
        scalar_t cube_sz;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
            }

            int64_t node_id;
            int32_t* data_idx_ptr = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz, tree.weight_accum != nullptr ? &node_id : nullptr);
            
            scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];

            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);
            
            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + opt.step_size;
            const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;
            if (sigma > opt.sigma_thresh) {
                transform_coord_world<scalar_t>(pos, tree.offset, tree.scaling);
                depth_out[0] = delta_scale * t;
                return;
            }
            t += delta_t;
        }
    }
}

template <typename scalar_t>
__global__ void motion_render_ray_kernel(
        PackedTreeSpec<scalar_t> tree,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
        torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
            out,
        torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
            depth_out,
        torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
            hit_point_out,
        torch::PackedTensorAccessor32<int64_t, 2, torch::RestrictPtrTraits>
            data_idx_out) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    motion_trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        out[tid], 
        depth_out[tid],
        hit_point_out[tid],
        data_idx_out[tid]
    );
}


template <typename scalar_t>
__global__ void depth_render_ray_kernel(
        PackedTreeSpec<scalar_t> tree,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
        torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
            depth_out) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    depth_trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        depth_out[tid]
    );
}


template <typename scalar_t>
__device__ __inline__ void motion_feature_trace_ray(
        PackedTreeSpec<scalar_t>& __restrict__ tree,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.features.size(1);
    const int joint_num = tree.joint_features.size(0);
    const int joint_feature_dim = tree.joint_features.size(1);
    const int binding_bone_num = tree.joint_index.size(1);
    const int out_data_dim = out.size(0);
    
    // printf("const ints: %d, %d, %d, %d, %d\n",  data_dim, joint_num, joint_feature_dim, out_data_dim, binding_bone_num);

    const int tmp_data_dim = 32;

    #pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
            // out[j] = opt.background_brightness;
        }
        return;
    } else {
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        scalar_t pos[3];
        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        scalar_t cube_sz;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
            }

            int64_t node_id;
            int32_t* data_idx_ptr = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz, tree.weight_accum != nullptr ? &node_id : nullptr);
            
            scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];
            scalar_t* skinning_weight_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.skinning_weights[*data_idx_ptr][0];
            int32_t* joint_index_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.joint_index[*data_idx_ptr][0];

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);
            
            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + opt.step_size;
            const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;
            // printf("data_idx: %d, cub_size: %f, feature_size: %d, sigma: %f, delta: %f, data_dim: %d\n", *data_idx_ptr, cube_sz, tree.features.size(0), sigma, delta_t, data_dim);
            // printf("motion features: %d, dim: %d\n", binding_bone_num, joint_feature_dim);
            if (sigma > opt.sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                const scalar_t weight = light_intensity * (1.f - att);

                scalar_t pos_joint_feature[tmp_data_dim];
                for (int j = 0; j != out_data_dim ; ++j) pos_joint_feature[j] = 0.f;
                for (int j = 0; j != binding_bone_num; ++j) {
                    if (skinning_weight_val[j] > 0) 
                        #pragma unroll
                        for (int k = 0; k != out_data_dim; ++k) pos_joint_feature[k] += skinning_weight_val[j] * tree.joint_features[joint_index_val[j]][k];                      
                }
                
                for (int j = 0; j != out_data_dim; ++j) {
                    out[j] += weight / ( 1.0 + expf(-pos_joint_feature[j]));
                }

                light_intensity *= att;

                if (light_intensity <= opt.stop_thresh) {
                    // Full opacity, stop
                    scalar_t scale = 1.0 / (1.0 - light_intensity);
                    for (int j = 0; j != out_data_dim; ++j) out[j] *= scale;
                    return;
                }
            }
            t += delta_t;
        }
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] += light_intensity * opt.background_brightness;
        }
    }
}

template <typename scalar_t>
__device__ __inline__ void motion_feature_trace_ray_backward(
    PackedTreeSpec<scalar_t>& __restrict__ tree,
    const torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t>
        grad_output,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        grad_data_out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.features.size(1);
    const int joint_num = tree.joint_features.size(0);
    const int joint_feature_dim = tree.joint_features.size(1);
    const int binding_bone_num = tree.joint_index.size(1);
    const int out_data_dim = grad_output.size(0);

    const int tmp_data_dim = 32;

    #pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        scalar_t pos[3];
        // PASS 1
        {
            scalar_t light_intensity = 1.f, t = tmin, cube_sz;
            while (t < tmax) {
                for (int j = 0; j < 3; ++j) pos[j] = ray.origin[j] + t * ray.dir[j];

                int32_t* data_idx_ptr = query_single_from_root<scalar_t>(tree.data, tree.child, pos, &cube_sz);
                scalar_t* tree_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.features[*data_idx_ptr][0];
                scalar_t* skinning_weight_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.skinning_weights[*data_idx_ptr][0];
                int32_t* joint_index_val = *data_idx_ptr >= tree.features.size(0) ? nullptr : &tree.joint_index[*data_idx_ptr][0];

                scalar_t att;
                scalar_t subcube_tmin, subcube_tmax;
                _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

                const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
                const scalar_t delta_t = t_subcube + opt.step_size;
                const scalar_t sigma = tree_val != nullptr ? tree_val[data_dim - 1] : 0.0;

                if (sigma > 0.0) {
                    scalar_t grad_tmp[tmp_data_dim];

                    att = expf(-delta_t * sigma * delta_scale);
                    const scalar_t weight = light_intensity * (1.f - att);

                    scalar_t pos_joint_feature[tmp_data_dim];
                    for (int j = 0; j != tmp_data_dim ; ++j) pos_joint_feature[j] = 0.f;
                    for (int j = 0; j != binding_bone_num ; ++j) {
                        if (skinning_weight_val[j] > 0) 
                            for (int k = 0; k != out_data_dim; ++k) pos_joint_feature[k] += skinning_weight_val[j] * tree.joint_features[joint_index_val[j]][k];                      
                    }

                    for (int j = 0; j != out_data_dim; ++j) {
                        const scalar_t sigmoid = 1.0 / (1.0 + expf(-pos_joint_feature[j]));
                        const scalar_t toadd = weight * sigmoid * (
                                    1.f - sigmoid) * grad_output[j];
                        grad_tmp[j] += toadd;
                    }

                    for (int j = 0; j != binding_bone_num ; ++j) {
                        if (skinning_weight_val[j] > 0) 
                            for (int k = 0; k != out_data_dim; ++k) atomicAdd(&grad_data_out[joint_index_val[j]][k], skinning_weight_val[j] * grad_tmp[j]);                      
                    }

                    light_intensity *= att;
                }
                t += delta_t;
            }
        }
    }
}

template <typename scalar_t>
__global__ void moition_feature_render_ray_kernel(
        PackedTreeSpec<scalar_t> tree,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        out) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    // printf("tid: %d\n", tid);
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    motion_feature_trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        out[tid]);
}


template <typename scalar_t>
__global__ void moition_feature_render_ray_backward_kernel(
    PackedTreeSpec<scalar_t> tree,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        grad_output,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        grad_data_out
        ) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    motion_feature_trace_ray_backward<scalar_t>(
        tree,
        grad_output[tid],
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        grad_data_out);
}

template <typename scalar_t>
__global__ void opacity_render_ray_kernel(
        PackedTreeSpec<scalar_t> tree,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        out) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    // printf("tid: %d\n", tid);
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    opacity_trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        out[tid]);
}

template <typename scalar_t>
__global__ void opacity_render_ray_backward_kernel(
    PackedTreeSpec<scalar_t> tree,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        grad_output,
        PackedRaysSpec<scalar_t> rays,
        RenderOptions opt,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        grad_data_out
        ) {
    CUDA_GET_THREAD_ID(tid, rays.origins.size(0));
    scalar_t origin[3] = {rays.origins[tid][0], rays.origins[tid][1], rays.origins[tid][2]};
    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    scalar_t dir[3] = {rays.dirs[tid][0], rays.dirs[tid][1], rays.dirs[tid][2]};
    opacity_trace_ray_backward<scalar_t>(
        tree,
        grad_output[tid],
        SingleRaySpec<scalar_t>{origin, dir, &rays.vdirs[tid][0]},
        opt,
        grad_data_out);
}



template <typename scalar_t>
__device__ __inline__ void cam2world_ray(
    int ix, int iy,
    scalar_t* dir,
    scalar_t* origin,
    const PackedCameraSpec<scalar_t>& __restrict__ cam) {
    scalar_t x = (ix - 0.5 * cam.width) / cam.fx;
    scalar_t y = -(iy - 0.5 * cam.height) / cam.fy;
    scalar_t z = sqrtf(x * x + y * y + 1.0);
    x /= z; y /= z; z = -1.0f / z;
    dir[0] = cam.c2w[0][0] * x + cam.c2w[0][1] * y + cam.c2w[0][2] * z;
    dir[1] = cam.c2w[1][0] * x + cam.c2w[1][1] * y + cam.c2w[1][2] * z;
    dir[2] = cam.c2w[2][0] * x + cam.c2w[2][1] * y + cam.c2w[2][2] * z;
    origin[0] = cam.c2w[0][3]; origin[1] = cam.c2w[1][3]; origin[2] = cam.c2w[2][3];
}


template <typename scalar_t>
__host__ __device__ __inline__ static void maybe_world2ndc(
        RenderOptions& __restrict__ opt,
        scalar_t* __restrict__ dir,
        scalar_t* __restrict__ cen, scalar_t near = 1.f) {
    if (opt.ndc_width < 0)
        return;
    scalar_t t = -(near + cen[2]) / dir[2];
    for (int i = 0; i < 3; ++i) {
        cen[i] = cen[i] + t * dir[i];
    }

    dir[0] = -((2 * opt.ndc_focal) / opt.ndc_width) * (dir[0] / dir[2] - cen[0] / cen[2]);
    dir[1] = -((2 * opt.ndc_focal) / opt.ndc_height) * (dir[1] / dir[2] - cen[1] / cen[2]);
    dir[2] = -2 * near / cen[2];

    cen[0] = -((2 * opt.ndc_focal) / opt.ndc_width) * (cen[0] / cen[2]);
    cen[1] = -((2 * opt.ndc_focal) / opt.ndc_height) * (cen[1] / cen[2]);
    cen[2] = 1 + 2 * near / cen[2];

    _normalize(dir);
}


template <typename scalar_t>
__global__ void render_image_kernel(
    PackedTreeSpec<scalar_t> tree,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        out) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};
    maybe_world2ndc(opt, dir, origin);

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, vdir},
        opt,
        out[iy][ix]);
}

template <typename scalar_t>
__global__ void render_image_backward_kernel(
    PackedTreeSpec<scalar_t> tree,
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grad_output,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    torch::PackedTensorAccessor64<scalar_t, 2, torch::RestrictPtrTraits>
        grad_data_out) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};
    maybe_world2ndc(opt, dir, origin);

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    trace_ray_backward<scalar_t>(
        tree,
        grad_output[iy][ix],
        SingleRaySpec<scalar_t>{origin, dir, vdir},
        opt,
        grad_data_out);
}

template <typename scalar_t>
__device__ __inline__ void grid_trace_ray(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        data,
        const scalar_t* __restrict__ origin,
        const scalar_t* __restrict__ dir,
        const scalar_t* __restrict__ vdir,
        scalar_t step_size,
        scalar_t delta_scale,
        scalar_t sigma_thresh,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_weight,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_hit) {
    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int reso = data.size(0);
    scalar_t* grid_weight_val = grid_weight.data();
    scalar_t* grid_hit_val = grid_hit.data();

#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (dir[i] + 1e-9);
    }
    _dda_unit(origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        return;
    } else {
        scalar_t pos[3];

        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        scalar_t cube_sz = reso;
        int32_t u, v, w, node_id;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = origin[j] + t * dir[j];
            }

            clamp_coord<scalar_t>(pos);
            pos[0] *= reso;
            pos[1] *= reso;
            pos[2] *= reso;
            u = floor(pos[0]);
            v = floor(pos[1]);
            w = floor(pos[2]);
            pos[0] -= u;
            pos[1] -= v;
            pos[2] -= w;
            node_id = u * reso * reso + v * reso + w;

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + step_size;
            const scalar_t sigma = data[u][v][w];
            if (sigma > sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                const scalar_t weight = light_intensity * (1.f - att);
                light_intensity *= att;

                atomicMax(&grid_weight_val[node_id], weight);
                atomicAdd(&grid_hit_val[node_id], (scalar_t) 1.0);
            }
            t += delta_t;
        }
    }
}

template <typename scalar_t>
__global__ void grid_weight_render_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        data,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    const scalar_t* __restrict__ offset,
    const scalar_t* __restrict__ scaling,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_weight,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        grid_hit) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};
    maybe_world2ndc(opt, dir, origin);

    transform_coord<scalar_t>(origin, offset, scaling);
    const scalar_t delta_scale = _get_delta_scale(scaling, dir);
    grid_trace_ray<scalar_t>(
        data,
        origin,
        dir,
        vdir,
        opt.step_size,
        delta_scale,
        opt.sigma_thresh,
        grid_weight,
        grid_hit);
}



}  // namespace device


// Compute RGB output dimension from input dimension & SH degree
__host__ int get_out_data_dim(int format, int basis_dim, int in_data_dim) {
    if (format != FORMAT_RGBA) {
        return (in_data_dim - 1) / basis_dim + 1;
    } else {
        return in_data_dim;
    }
}

}  // namespace

torch::Tensor volume_render(TreeSpec& tree, RaysSpec& rays, RenderOptions& opt) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.features);
    const auto Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.features.size(1));
    torch::Tensor result = torch::zeros({Q, out_data_dim}, rays.origins.options());
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::render_ray_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, rays, opt,
                    result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor volume_render_image(TreeSpec& tree, CameraSpec& cam, RenderOptions& opt) {
    tree.check();
    cam.check();
    DEVICE_GUARD(tree.features);
    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.features.size(1));
    torch::Tensor result = torch::zeros({cam.height, cam.width, out_data_dim},
            tree.data.options());

    AT_DISPATCH_FLOATING_TYPES(tree.data.type(), __FUNCTION__, [&] {
            device::render_image_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, cam, opt,
                    result.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor volume_render_backward(
    TreeSpec& tree, RaysSpec& rays,
    RenderOptions& opt,
    torch::Tensor grad_output) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.features);

    const int Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.features.size(1));
    torch::Tensor result = torch::zeros_like(tree.features);
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::render_ray_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                tree,
                grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                rays,
                opt,
                result.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor volume_render_image_backward(TreeSpec& tree, CameraSpec& cam,
                                           RenderOptions& opt,
                                           torch::Tensor grad_output) {
    tree.check();
    cam.check();
    DEVICE_GUARD(tree.features);

    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.features.size(1));
    torch::Tensor result = torch::zeros_like(tree.features);

    AT_DISPATCH_FLOATING_TYPES(tree.data.type(), __FUNCTION__, [&] {
            device::render_image_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                tree,
                grad_output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                cam,
                opt,
                result.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

std::vector<torch::Tensor> grid_weight_render(
    torch::Tensor data, CameraSpec& cam, RenderOptions& opt,
    torch::Tensor offset, torch::Tensor scaling) {
    cam.check();
    DEVICE_GUARD(data);
    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    torch::Tensor grid_weight = torch::zeros_like(data);
    torch::Tensor grid_hit = torch::zeros_like(data);

    AT_DISPATCH_FLOATING_TYPES(data.type(), __FUNCTION__, [&] {
            device::grid_weight_render_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                data.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                cam,
                opt,
                offset.data<scalar_t>(),
                scaling.data<scalar_t>(),
                grid_weight.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                grid_hit.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return {grid_weight, grid_hit};
}

std::vector<torch::Tensor> motion_render(TreeSpec& tree, RaysSpec& rays, RenderOptions& opt) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.extra_data);
    const auto Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int motion_feature_dim = tree.extra_data.size(0);
    torch::Tensor result = torch::zeros({Q, motion_feature_dim}, rays.origins.options());
    torch::Tensor depth = torch::zeros({Q, 1}, rays.origins.options());
    torch::Tensor hit_point = torch::zeros({Q, 3}, rays.origins.options());
    torch::Tensor data_idx = torch::zeros({Q, 1}, rays.origins.options()).to(torch::kInt64);
    
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::motion_render_ray_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, rays, opt,
                    result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    depth.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    hit_point.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                    data_idx.packed_accessor32<int64_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return {result, depth, hit_point, data_idx};
}

torch::Tensor render_depth(TreeSpec& tree, RaysSpec& rays, RenderOptions& opt) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.extra_data);
    const auto Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    torch::Tensor depth = torch::zeros({Q, 1}, rays.origins.options());
    
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::depth_render_ray_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, rays, opt,
                    depth.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return depth;
}

torch::Tensor motion_feature_render(TreeSpec& tree, RaysSpec& rays, RenderOptions& opt) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.joint_features);
    const auto Q = rays.origins.size(0);

    auto_cuda_threads();
    // cuda_n_threads = 256;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = tree.joint_features.size(1);
    torch::Tensor result = torch::zeros({Q, out_data_dim}, rays.origins.options());
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::moition_feature_render_ray_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, rays, opt,
                    result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}


torch::Tensor motion_feature_render_backward(
    TreeSpec& tree, RaysSpec& rays,
    RenderOptions& opt,
    torch::Tensor grad_output) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.joint_features);

    const int Q = rays.origins.size(0);

    auto_cuda_threads();

    // cuda_n_threads = 256;
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = tree.joint_features.size(1);
    torch::Tensor result = torch::zeros_like(tree.joint_features);
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::moition_feature_render_ray_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                tree,
                grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                rays,
                opt,
                result.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor opacity_render(TreeSpec& tree, RaysSpec& rays, RenderOptions& opt) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.features);
    const auto Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = 1;
    torch::Tensor result = torch::zeros({Q, out_data_dim}, rays.origins.options());
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::opacity_render_ray_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, rays, opt,
                    result.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}

torch::Tensor opacity_render_backward(
    TreeSpec& tree, RaysSpec& rays,
    RenderOptions& opt,
    torch::Tensor grad_output) {
    tree.check();
    rays.check();
    DEVICE_GUARD(tree.features);

    const int Q = rays.origins.size(0);

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    torch::Tensor result = torch::zeros_like(tree.features);
    AT_DISPATCH_FLOATING_TYPES(rays.origins.type(), __FUNCTION__, [&] {
            device::render_ray_backward_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                tree,
                grad_output.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                rays,
                opt,
                result.packed_accessor64<scalar_t, 2, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}
